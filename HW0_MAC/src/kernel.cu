#include "hip/hip_runtime.h"
// CIS565 CUDA Checker: A simple CUDA hello-world style program for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cutil_math.h>
#include "kernel.h"
#include <iostream>

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
    exit(EXIT_FAILURE); 
  }
} 

//Kernel that writes the image to the OpenGL PBO directly. 
__global__ void createVersionVisualization(uchar4* PBOpos, int width, int height, int major, int minor){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * width);
  
  if(x<=width && y<=height){ 
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = 0;     
      PBOpos[index].y = 0;
      PBOpos[index].z = 0;

      if(y<height/2){
        if(major==1){
          PBOpos[index].x = 255;  
        }else if(major==2){
          PBOpos[index].y = 255;  
        }else if(major==3){
          PBOpos[index].z = 255;  
        }
      }else{
        if(minor==0){
          PBOpos[index].x = 255;  
        }else if(minor==1){
          PBOpos[index].y = 255;  
        }else if(minor==2){
          PBOpos[index].z = 255;  
        }else if(minor==3){
          PBOpos[index].x = 255;  
          PBOpos[index].y = 255;  
        }else if(minor==5){
          PBOpos[index].z = 255;  
          PBOpos[index].y = 255;  
        }
      }
  }
}

// Wrapper for the __global__ call that sets up the kernel calls and does a ton of memory management
void cudaKernel(uchar4* PBOpos, int width, int height, int major, int minor){
  
  // set up crucial magic
  int tileSize = 16;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(width/float(tileSize)), (int)ceil(height/float(tileSize)));
  
  //kernel launches
  createVersionVisualization<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, width, height, major, minor);
  // make certain the kernel has completed 
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}
